#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string>
#include <cstdio>

#ifndef PTX_DIR
#define PTX_DIR "."
#endif

#define CUDA_CHECK(call)                                                                \
    do {                                                                                \
        hipError_t res = (call);                                                          \
        if (res != hipSuccess) {                                                      \
            const char *errStr;                                                         \
            hipDrvGetErrorString(res, &errStr);                                             \
            fprintf(stderr, "CUDA error at %s:%d -- %s\n", __FILE__, __LINE__, errStr); \
            exit(EXIT_FAILURE);                                                         \
        }                                                                               \
    } while (0)

#define WARP_SIZE 32

void cpu_prefix_sum(float *output, int N, float val) {
    for (int i = 0; i < N; i++) {
        output[i] = val * (i + 1);
    }
}

int main() {
    CUDA_CHECK(hipInit(0));
    hipDevice_t cuDevice;
    CUDA_CHECK(hipDeviceGet(&cuDevice, 0));
    hipCtx_t cuContext;
    CUDA_CHECK(hipCtxCreate(&cuContext, 0, cuDevice));

    std::string ptx_path = std::string(PTX_DIR) + "/prefix_sum.ptx";
    FILE* ptx_file = fopen(ptx_path.c_str(), "rb");
    if (!ptx_file) {
        printf("Failed to open PTX file: %s\n", ptx_path.c_str());
        return 1;
    }
    
    fseek(ptx_file, 0, SEEK_END);
    size_t size = ftell(ptx_file);
    fseek(ptx_file, 0, SEEK_SET);
    
    std::string ptx_content;
    ptx_content.resize(size);
    fread(&ptx_content[0], 1, size, ptx_file);
    fclose(ptx_file);

    hipModule_t cuModule;
    CUDA_CHECK(hipModuleLoadData(&cuModule, ptx_content.c_str()));

    hipFunction_t kernel;
    CUDA_CHECK(hipModuleGetFunction(&kernel, cuModule, "warpPrefixSum"));

    hipDeviceptr_t d_output;
    size_t output_size = WARP_SIZE * sizeof(float);
    CUDA_CHECK(hipMalloc(&d_output, output_size));

    float inputVal = 1.0f;
    void *kernel_params[2];
    kernel_params[0] = &inputVal;
    kernel_params[1] = &d_output;

    CUDA_CHECK(hipModuleLaunchKernel(kernel,
                              1, 1, 1,
                              WARP_SIZE, 1, 1,
                              0,
                              0,
                              kernel_params,
                              0));
    CUDA_CHECK(hipCtxSynchronize());

    float h_output[WARP_SIZE] = {0};
    CUDA_CHECK(hipMemcpyDtoH(h_output, d_output, output_size));

    float expected[WARP_SIZE] = {0};
    cpu_prefix_sum(expected, WARP_SIZE, inputVal);

    int correct = 1;
    for (int i = 0; i < WARP_SIZE; i++) {
        if (fabs(h_output[i] - expected[i]) > 1e-6) {
            correct = 0;
            printf("Mismatch at index %d: GPU = %f, CPU = %f\n", i, h_output[i], expected[i]);
        }
    }
    if (correct) {
        printf("Test PASSED! GPU results match CPU results.\n");
    } else {
        printf("Test FAILED!\n");
    }

    printf("\nWarp prefix sum results (GPU):\n");
    for (int i = 0; i < WARP_SIZE; i++) {
        printf("Thread %2d: %f\n", i, h_output[i]);
    }

    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipModuleUnload(cuModule));
    CUDA_CHECK(hipCtxDestroy(cuContext));

    return 0;
}
