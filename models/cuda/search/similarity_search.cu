#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>
#include <thrust/scan.h>
#include <iostream>
#include <random>
#include <chrono>
#include "../utils.h"

#define DEBUG 0
#define PROFILING 0

struct SimilarityResult {
    int document_index;
    float similarity_score;
};

SimilarityResult similarity_search_cpu(
    const float *query_embeddings,
    const float *document_embeddings,
    const int num_queries,
    const int num_documents,
    const int embedding_dim,
    const float score_threshold = 0.75f
) {
    int most_similar_document_index = -1;
    float highest_similarity_score = 0.0f;

    for (int i = 0; i < num_queries; i++) {
        for (int j = 0; j < num_documents; j++) {
            float dot_product = 0.0f;
            float vec1_dp = 0.0f;
            float vec2_dp = 0.0f;

            for (int k = 0; k < embedding_dim; k++) {
                float q = query_embeddings[i * embedding_dim + k];
                float d = document_embeddings[j * embedding_dim + k];
                dot_product += q * d;
                vec1_dp += q * q;
                vec2_dp += d * d;
            }

            float similarity = (vec1_dp > 0.0f && vec2_dp > 0.0f) ? 
                               (dot_product * rsqrtf(vec1_dp) * rsqrtf(vec2_dp)) : 0.0f;

            if (similarity > highest_similarity_score && similarity >= score_threshold) {
                highest_similarity_score = similarity;
                most_similar_document_index = j;
            }
        }
    }

    return SimilarityResult{most_similar_document_index, highest_similarity_score};
}

__global__ void similarity_search_kernel(
    SimilarityResult *result,
    const float *query_embeddings, 
    const float *document_embeddings,
    const int num_queries,
    const int num_documents,
    const int embedding_dim,
    const float score_threshold
) {
    __shared__ SimilarityResult local_result;
    if (threadIdx.x == 0) {
        local_result.document_index = -1;
        local_result.similarity_score = -1.0f;
    }
    __syncthreads();

    int query_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (query_idx < num_queries) {
        for (int doc = 0; doc < num_documents; ++doc) {
            float dot_product = 0.0f;
            float vec1_dp = 0.0f;
            float vec2_dp = 0.0f;

            for (int i = 0; i < embedding_dim; ++i) {
                float q = query_embeddings[query_idx * embedding_dim + i];
                float d = document_embeddings[doc * embedding_dim + i];
                dot_product += q * d;
                vec1_dp += q * q; 
                vec2_dp += d * d;
            }

            float similarity = (vec1_dp > 0.0f && vec2_dp > 0.0f) ? 
                               (dot_product * rsqrtf(vec1_dp) * rsqrtf(vec2_dp)) : 0.0f;

            if (similarity >= score_threshold) {
                atomicMax((int*)&local_result.similarity_score, __float_as_int(similarity));
                if (similarity == __int_as_float(atomicMax((int*)&local_result.similarity_score, __float_as_int(similarity)))) {
                    atomicExch(&local_result.document_index, doc);
                }
            }
        }
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        atomicMax((int*)&result->similarity_score, __float_as_int(local_result.similarity_score));
        if (local_result.similarity_score == __int_as_float(atomicMax((int*)&result->similarity_score, __float_as_int(local_result.similarity_score)))) {
            atomicExch(&result->document_index, local_result.document_index);
        }
    }
}

void similarity_search(
    const float *query_embeddings,
    const float *document_embeddings,
    const int num_queries,
    const int num_documents,
    const int embedding_dim,
    const float score_threshold,
    SimilarityResult *result
) {
    CUDA_ASSERT(hipFree(0));

    int threads_per_block = 256;
    int blocks_per_grid = DIV_CEIL(num_queries, threads_per_block);

    hipStream_t stream;
    CUDA_ASSERT(hipStreamCreate(&stream));

    similarity_search_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(
        result,
        query_embeddings,
        document_embeddings,
        num_queries,
        num_documents,
        embedding_dim,
        score_threshold
    );

    CUDA_ASSERT(hipGetLastError());
    CUDA_ASSERT(hipStreamSynchronize(stream));
    CUDA_ASSERT(hipStreamDestroy(stream));
}

int main() {
    const int num_queries = 10;
    const int num_documents = 10;
    const int embedding_dim = 768;
    const float score_threshold = 0.75f;

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(0.0, 1.0);

    thrust::host_vector<float> h_query_embeddings(num_queries * embedding_dim);
    thrust::host_vector<float> h_document_embeddings(num_documents * embedding_dim);

    auto randomize_embeddings = [&]() {
        for (auto &val : h_query_embeddings) {
            val = dis(gen);
        }
        for (auto &val : h_document_embeddings) {
            val = dis(gen);
        }
    };

    thrust::device_vector<float> d_query_embeddings(num_queries * embedding_dim);
    thrust::device_vector<float> d_document_embeddings(num_documents * embedding_dim);
    thrust::device_vector<SimilarityResult> d_result(1);

    const int num_tests = 5;

    for (int test = 0; test < num_tests; ++test) {
        randomize_embeddings();
        thrust::copy(h_query_embeddings.begin(), h_query_embeddings.end(), d_query_embeddings.begin());
        thrust::copy(h_document_embeddings.begin(), h_document_embeddings.end(), d_document_embeddings.begin());

        SimilarityResult cpu_result = similarity_search_cpu(
            h_query_embeddings.data(),
            h_document_embeddings.data(),
            num_queries,
            num_documents,
            embedding_dim,
            score_threshold
        );

        similarity_search(
            thrust::raw_pointer_cast(d_query_embeddings.data()),
            thrust::raw_pointer_cast(d_document_embeddings.data()),
            num_queries,
            num_documents,
            embedding_dim,
            score_threshold,
            thrust::raw_pointer_cast(d_result.data())
        );

        SimilarityResult gpu_result;
        thrust::copy(d_result.begin(), d_result.end(), &gpu_result);

        if (DEBUG) {
            if (gpu_result.document_index != cpu_result.document_index ||
                fabs(gpu_result.similarity_score - cpu_result.similarity_score) > 1e-5) {
                std::cout << "Mismatch: "
                        << "CPU index = " << cpu_result.document_index
                        << ", GPU index = " << gpu_result.document_index
                        << "; CPU score = " << cpu_result.similarity_score
                        << ", GPU score = " << gpu_result.similarity_score << std::endl;
            } else {
                std::cout << "GPU result matches CPU result." << std::endl;
                std::cout << "Document index: " << gpu_result.document_index 
                        << ", Similarity score: " << gpu_result.similarity_score << std::endl;
            }
        }
    }

    if (PROFILING) {
        auto start_cpu = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < 100; ++i) {
            randomize_embeddings();
            similarity_search_cpu(
                h_query_embeddings.data(),
                h_document_embeddings.data(),
                num_queries,
                num_documents,
                embedding_dim,
                score_threshold
            );
        }
        auto end_cpu = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> cpu_duration = end_cpu - start_cpu;
        std::cout << "CPU similarity search time (100 iterations): " 
                  << cpu_duration.count() / 100 << " seconds per iteration" << std::endl;

        auto start_gpu = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < 100; ++i) {
            randomize_embeddings();
            thrust::copy(h_query_embeddings.begin(), h_query_embeddings.end(), d_query_embeddings.begin());
            thrust::copy(h_document_embeddings.begin(), h_document_embeddings.end(), d_document_embeddings.begin());
            
            similarity_search(
                thrust::raw_pointer_cast(d_query_embeddings.data()),
                thrust::raw_pointer_cast(d_document_embeddings.data()),
                num_queries,
                num_documents,
                embedding_dim,
                score_threshold,
                thrust::raw_pointer_cast(d_result.data())
            );
        }
        auto end_gpu = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> gpu_duration = end_gpu - start_gpu;
        std::cout << "GPU similarity search time (100 iterations): " 
                  << gpu_duration.count() / 100 << " seconds per iteration" << std::endl;
    }

    return 0;
}
